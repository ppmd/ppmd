#include "hip/hip_runtime.h"
        //source

        __constant__ int d_n;
        //__constant__ int d_nl;
        __constant__ int d_nc;

        __constant__ double _icel0;
        __constant__ double _icel1;
        __constant__ double _icel2;

        __constant__ double _b0;
        __constant__ double _b2;
        __constant__ double _b4;
        __constant__ double _b1;
        __constant__ double _b3;
        __constant__ double _b5;

        __constant__ double _ca0;
        __constant__ double _ca1;
        __constant__ double _ca2;

        __global__ void d_LayerSort(int* __restrict__ d_pl,
                                    int* __restrict__ d_crl,
                                    int* __restrict__ d_ccc,
                                    const double* __restrict__ d_p
        ){

        const int _ix = threadIdx.x + blockIdx.x*blockDim.x;
        if (_ix < d_n){

            int C0 = 1 + __double2int_rz(( d_p[_ix*3]    - _b0 )*_icel0);
            int C1 = 1 + __double2int_rz(( d_p[_ix*3+1]  - _b2 )*_icel1);
            int C2 = 1 + __double2int_rz(( d_p[_ix*3+2]  - _b4 )*_icel2);

            if ( (C0 > (_ca0-2)) && (d_p[_ix*3]   <= _b1 )) {C0 = _ca0-2;}
            if ( (C1 > (_ca1-2)) && (d_p[_ix*3+1] <= _b3 )) {C1 = _ca1-2;}
            if ( (C2 > (_ca2-2)) && (d_p[_ix*3+2] <= _b5 )) {C2 = _ca2-2;}

            const int val = (C2*_ca1 + C1)*_ca0 + C0;

            d_crl[_ix] = val;
            //old=atomicAdd(address, new);

            d_pl[_ix] = atomicAdd(&d_ccc[val], (int)1);

        }
        return;
        }

        __global__ void d_MaxLayers(const int* __restrict__ d_ccc, int * nl_out){
        
        const int _ix = threadIdx.x + blockIdx.x*blockDim.x;
        int val = 0;

        if (_ix < d_nc){
            val = d_ccc[_ix];
        }
        
        for (int offset = warpSize/2; offset > 0; offset /=2){
            //val = fmaxf(val, __shfl_down(val,offset));
            int tmp = __shfl_down(val,offset);
            //val = (val > tmp) ? val : tmp;
            //asm("max.s32 %0, %1, %2;" : "=r"(val) : "r"(val), "r"(tmp));
            val = max(val,tmp);
        }
        
        if ((int)(threadIdx.x & (warpSize - 1)) == 0){
            atomicMax(nl_out, val);
        }
        
        return;
        }

        __global__ void d_PopulateMatrix(const int d_nl,
                                         const int* __restrict__ d_pl,
                                         const int* __restrict__ d_crl,
                                         int* __restrict__ d_M
        ){

        const int _ix = threadIdx.x + blockIdx.x*blockDim.x;
        if (_ix < d_n){
            
            d_M[ d_crl[_ix]*d_nl + d_pl[_ix]  ] = _ix;
        }
        return;
        }


        int LayerSort(const int f_MPI_COMM,
                     const int MPI_FLAG,
                     const int blocksize[3],
                     const int threadsize[3],
                     const int blocksize2[3],
                     const int threadsize2[3],
                     const int n,
                     const int nc,
                     int* nl,
                     int* n_cells,
                     int* __restrict__ d_pl,
                     int* __restrict__ d_crl,
                     int* __restrict__ d_ccc,
                     int** __restrict__ d_M,
                     const int* __restrict__ h_ca,
                     const double* __restrict__ h_b,
                     const double* __restrict__ h_cel,
                     const double* __restrict__ d_p
                     ){
            int err = 0;

            MPI_Comm MPI_COMM;
            if (MPI_FLAG > 0){
                MPI_COMM = MPI_Comm_f2c(f_MPI_COMM);
            }

            const double _hicel0 = 1.0/h_cel[0];
            const double _hicel1 = 1.0/h_cel[1];
            const double _hicel2 = 1.0/h_cel[2];

            const double _hb0 = h_b[0];
            const double _hb2 = h_b[2];
            const double _hb4 = h_b[4];
            const double _hb1 = h_b[1];
            const double _hb3 = h_b[3];
            const double _hb5 = h_b[5];

            const double _hca0 = h_ca[0];
            const double _hca1 = h_ca[1];
            const double _hca2 = h_ca[2];

            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_icel0), &_hicel0, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_icel1), &_hicel1, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_icel2), &_hicel2, sizeof(double)));

            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_b0), &_hb0, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_b2), &_hb2, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_b4), &_hb4, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_b1), &_hb1, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_b3), &_hb3, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_b5), &_hb5, sizeof(double)));

            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_ca0), &_hca0, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_ca1), &_hca1, sizeof(double)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_ca2), &_hca2, sizeof(double)));


            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_n), &n, sizeof(n)));
            checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_nc), &nc, sizeof(nc)));
            //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_nl), nl, sizeof(*nl)));

            dim3 bs; bs.x = blocksize[0]; bs.y = blocksize[1]; bs.z = blocksize[2];
            dim3 ts; ts.x = threadsize[0]; ts.y = threadsize[1]; ts.z = threadsize[2];

            if (n > 0){
                d_LayerSort<<<bs,ts>>>(d_pl, d_crl, d_ccc, d_p);
            }

            checkCudaErrors(hipDeviceSynchronize());
            getLastCudaError(" d_LayerSort Execution failed. \n");
            
            // bit of global memory for maximum number of layers.
            int * d_nl; hipMalloc((void**)&d_nl, sizeof(int));
            hipMemcpy(d_nl, nl, sizeof(int), hipMemcpyHostToDevice);

            dim3 bs2; bs2.x = blocksize2[0]; bs2.y = blocksize2[1]; bs2.z = blocksize2[2];
            dim3 ts2; ts2.x = threadsize2[0]; ts2.y = threadsize2[1]; ts2.z = threadsize2[2];
            

            d_MaxLayers<<<bs2,ts2>>>(d_ccc, d_nl);
            int old_nl = *nl;
            checkCudaErrors(hipDeviceSynchronize());
            getLastCudaError(" d_MaxLayers Execution failed. \n");

            int tnl = 0;
            hipMemcpy(&tnl, d_nl, sizeof(int), hipMemcpyDeviceToHost);

            if(MPI_FLAG > 0){
                MPI_Allreduce(&tnl, nl, 1, MPI_INT, MPI_MAX, MPI_COMM);
            } else {
                *nl = tnl;
            }

            if ((*nl)*(*n_cells)>old_nl*(*n_cells)){
            //need to resize.
                hipFree(*d_M);
                //printf("new number of layers = %d, number of cells %d \n", *nl, *n_cells);
                checkCudaErrors(hipMalloc((void**)d_M, (*nl)*(*n_cells)*sizeof(int)));

                /*
                printf("new pointer %ld \n", (long)(*d_M));
                int tmp;
                int err=hipMemcpy(&tmp, *d_M + 511, sizeof(int), hipMemcpyDeviceToHost);
                printf("err %d, tmp %d \n", err, tmp);
                */

            }

            //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_nl), nl, sizeof(*nl)));
            if (n > 0){
                d_PopulateMatrix<<<bs,ts>>>(*nl, d_pl, d_crl, *d_M);
            }
            checkCudaErrors(hipDeviceSynchronize());

            return err;
        }


        // ---------- realloc matrix code --------------

        __global__ void copy_matrix_cols_kernel(
            const int d_n,
            const int d_old_ncol,
            const int d_new_ncol,
            const int d_nrow,
            const int * __restrict__ d_old_ptr,
            int * __restrict__ d_new_ptr
        ){
            const int ix = threadIdx.x + blockIdx.x * blockDim.x;
            if (ix<d_n){
                    const int row = ix/d_old_ncol;
                    const int col = ix % d_old_ncol;
                    const int val = d_old_ptr[row*d_old_ncol + col];
                    d_new_ptr[row*d_new_ncol + col] = val;
            }
            return;
        }

        int copy_matrix_cols(
            const int h_old_ncol,
            const int h_new_ncol,
            const int h_nrow,
            const int * __restrict__ d_old_ptr,
            int * __restrict__ d_new_ptr
        ){
            hipError_t err;
            dim3 bs, ts;
            const int h_n = h_old_ncol*h_nrow;
            err = cudaCreateLaunchArgs(h_n, 1024, &bs, &ts);
            if(err>0){return err;}
            copy_matrix_cols_kernel<<<bs,ts>>>(h_n,
                                               h_old_ncol,
                                               h_new_ncol,
                                               h_nrow,
                                               d_old_ptr,
                                               d_new_ptr);
            err = hipDeviceSynchronize();
            return err;

        }




        