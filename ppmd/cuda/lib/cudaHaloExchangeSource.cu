#include "hip/hip_runtime.h"


__global__ void cudaPackParticleDat(
    const int d_pos_flag,
    const int d_n,
    const int d_cccmax,
    const int d_occ_m_stride,
    const int d_offset,
    const int * __restrict__ d_b_indices,
    const int * __restrict__ d_occ_matrix,
    const int * __restrict__ d_ccc,
    const int * __restrict__ d_b_scan,
    const double* __restrict__ d_shift,
    const %(DTYPE)s * __restrict__ d_ptr,
    %(DTYPE)s * __restrict__ d_buffer
){

    const int ncomp = %(NCOMP)s;
    const int ix = threadIdx.x + blockIdx.x*blockDim.x;
    if (ix < (d_n*ncomp)){

        // cell index offset
        const int cio = ix / (ncomp * d_cccmax);
        // cell index
        const int cid = d_b_indices[cio];
        // particle layer in cell.
        const int pil = (ix/ncomp) %% d_cccmax;

        if (pil < d_ccc[cid]) {
            // particle component
            const int comp = ix %% ncomp;

            // get particle index.
            const int pid = ncomp * d_occ_matrix[d_occ_m_stride * cid + pil] + comp;

            /*
            if (cid==511){
            printf("(cid, stride, pil) %%d, %%d, %%d :ix %%d + %%d\n",
             cid,
             d_occ_m_stride,
             pil,
             d_occ_matrix[d_occ_m_stride * cid + pil],
             comp);
            }
            */


            // compute buffer index
            const int bid = ncomp * (d_b_scan[cio] - d_offset + pil) + comp;
            //const int bid = 0;

            //printf("\t(bid, pid, val, shift, d_n*ncomp) %%d, %%d, %%f, %%f, %%d\n", bid, pid, d_ptr[pid], d_shift[comp], d_n*ncomp);


            //if (bid > 599){printf("\t\t\tbid exceeded tmp (upper) dim");}
            //if (bid < 0){printf("\t\t\tbid exceeded tmp (lower) dim");}



            // copy data to buffer
            d_buffer[bid] = d_ptr[pid];

            // apply periodic boundary flag to packed particles
            if ( d_pos_flag == 1 ){
                d_buffer[bid] += d_shift[comp];
            }

        }

    }
    return;
}



int cudaHaloExchangePD(
    const int f_MPI_COMM,
    const int n_local,
    const int h_pos_flag,
    const int h_cccmax,
    const int h_occ_m_stride,
    const int* __restrict__ h_b_ind,
    const int* __restrict__ h_send_counts,
    const int* __restrict__ h_recv_counts,
    const int* __restrict__ SEND_RANKS,
    const int* __restrict__ RECV_RANKS,
    const int* __restrict__ d_b_indices,
    const int* __restrict__ d_occ_matrix,
    const int* __restrict__ d_ccc,
    const int* __restrict__ d_b_scan,
    const double* __restrict__ d_shift,
    %(DTYPE)s * __restrict__ d_ptr,
    %(DTYPE)s * __restrict__ d_buffer
){






    // get mpi comm and rank
    MPI_Comm MPI_COMM = MPI_Comm_f2c(f_MPI_COMM);
    int rank = -1; MPI_Comm_rank( MPI_COMM, &rank );
    MPI_Status MPI_STATUS;
    MPI_Request sr;
    MPI_Request rr;

    int DAT_END = n_local;
    int offset = 0;

    dim3 bs, ts;
    hipError_t err;


    //cout << "rank " << rank << endl;

    // ---
    /*
    int tmp;
    int err2 = hipMemcpy(&tmp, d_occ_matrix+511, sizeof(int), hipMemcpyDeviceToHost);
    cout << "511 BEFORE RUN: " << tmp << " err " << err2 << endl;

    if (rank==0){

    cout << f_MPI_COMM << endl;
    cout << n_local << endl;
    cout << h_pos_flag << endl;
    cout << h_cccmax << endl;
    cout << h_occ_m_stride << endl;

    for( int dir=0 ; dir<6 ; dir++ ){
    cout << "# " <<  dir << " ---- " << endl;
    cout << h_b_ind[dir] << endl;
    cout << SEND_RANKS[dir] << endl;
    cout << RECV_RANKS[dir] << endl;
    cout << h_send_counts[dir] << endl;
    cout << h_recv_counts[dir] << endl;
    }


    }

    */

    // ---


    for( int dir=0 ; dir<6 ; dir++ ){

        int b_s = h_b_ind[dir];
        const int cell_count = h_b_ind[dir+1] - b_s;


        const int scount = h_send_counts[dir];


        err = cudaCreateLaunchArgs(   cell_count*h_cccmax*%(NCOMP)s    , 256, &bs, &ts);
        if (err != hipSuccess) { return err; }

        /*
        if (rank==0){
        cout << "scount " << scount << " cell_count " << cell_count << endl;
        cout << "bs.x " << bs.x << " ts.x " << ts.x << endl;
        }
        */


        cudaPackParticleDat<<<bs,ts>>>(
            h_pos_flag,
            cell_count*h_cccmax,
            h_cccmax,
            h_occ_m_stride,
            offset,
            d_b_indices + b_s,
            d_occ_matrix,
            d_ccc,
            d_b_scan + b_s,
            d_shift + dir*3,
            d_ptr,
            d_buffer
        );

        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            //cout << "Error on cudaSync: " << rank << endl;
            return err;
         }


        if (( SEND_RANKS[dir] > -1 ) && ( h_send_counts[dir] > 0 ) ){
        MPI_Isend((void *) d_buffer, h_send_counts[dir] * %(NCOMP)s, %(MPI_DTYPE)s,
                 SEND_RANKS[dir], rank, MPI_COMM, &sr);
        }

        if (( RECV_RANKS[dir] > -1 ) && ( h_recv_counts[dir] > 0 ) ){
        MPI_Irecv((void *) &d_ptr[DAT_END * %(NCOMP)s], %(NCOMP)s * h_recv_counts[dir],
                  %(MPI_DTYPE)s, RECV_RANKS[dir], RECV_RANKS[dir], MPI_COMM, &rr);
        }


        // after send has completed move to next direction.
        if (( SEND_RANKS[dir] > -1 ) && ( h_send_counts[dir] > 0 ) ){
            MPI_Wait(&sr, &MPI_STATUS);
        }

        if (( RECV_RANKS[dir] > -1 ) && ( h_recv_counts[dir] > 0 ) ){
            MPI_Wait(&rr, &MPI_STATUS);
        }

        MPI_Barrier(MPI_COMM);
        DAT_END += h_recv_counts[dir];
        offset += h_send_counts[dir];

        /*
        if (rank==0){
        cout << "IL " << h_recv_counts[dir] << " " << h_send_counts[dir] << endl;
        }
        */

    }

    return err;
}