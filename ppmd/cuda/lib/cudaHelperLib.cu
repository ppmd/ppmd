#include "cudaHelperLib.h"
#include <iostream>

int cudaErrorCheck(int err){
    //checkCudaErrors((hipError_t) err);
    if (err != 0) {
        std::cout << hipGetErrorString((hipError_t) err) << std::endl;
    }
    return err;
}

int cudaCpyHostToDevice(void* dst, const void* src, size_t count){
    hipError_t err;
    err = hipMemcpy(dst,src,count,hipMemcpyHostToDevice);
    return (int) err;
}

int cudaCpyDeviceToHost(void* dst, const void* src, size_t count){
    hipError_t err;
    err = hipMemcpy(dst,
                src,
                count,
                hipMemcpyDeviceToHost);
    return (int) err;
}

int cudaCpyDeviceToDevice(void* dst, const void* src, size_t count){
    hipError_t err;
    err = hipMemcpy(dst,src,count,hipMemcpyDeviceToDevice);
    return (int) err;
}

int cudaHostRegisterWrapper(void* ptr, size_t size){
    hipError_t err;
    err = hipHostRegister(ptr, size, hipHostRegisterPortable);
    return (int) err;
}

int cudaHostUnregisterWrapper(void* ptr){
    hipError_t err;
    err = hipHostUnregister(ptr);
    return (int) err;
}
