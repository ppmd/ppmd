#include "cudaHelperLib.h"

int cudaErrorCheck(int err){
    checkCudaErrors((hipError_t) err);
    return err;
}

void cudaCpyHostToDevice(void* dst, const void* src, size_t count){
    checkCudaErrors(hipMemcpy(dst,src,count,hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}

void cudaCpyDeviceToHost(void* dst, const void* src, size_t count){
    checkCudaErrors(hipMemcpy(dst,
                src,
                count,
                hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}

void cudaCpyDeviceToDevice(void* dst, const void* src, size_t count){
    checkCudaErrors(hipMemcpy(dst,src,count,hipMemcpyDeviceToDevice));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}

void cudaHostRegisterWrapper(void* ptr, size_t size){
    checkCudaErrors(hipHostRegister(ptr, size, hipHostRegisterPortable));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}

void cudaHostUnregisterWrapper(void* ptr){
    checkCudaErrors(hipHostUnregister(ptr));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}
