#include "cudaHelperLib.h"

int cudaErrorCheck(int err){
    if (err != 0) {
        cout << hipGetErrorString((hipError_t) err) << endl;
    }
    return err;
}

int cudaCpyHostToDevice(void* dst, const void* src, size_t count){
    hipError_t err;
    err = hipMemcpy(dst,src,count,hipMemcpyHostToDevice);
    return (int) err;
}

int cudaCpyDeviceToHost(void* dst, const void* src, size_t count){
    hipError_t err;
    err = hipMemcpy(dst,
                src,
                count,
                hipMemcpyDeviceToHost);
    return (int) err;
}

int cudaCpyDeviceToDevice(void* dst, const void* src, size_t count){
    hipError_t err;
    err = hipMemcpy(dst,src,count,hipMemcpyDeviceToDevice);
    return (int) err;
}

int cudaHostRegisterWrapper(void* ptr, size_t size){
    hipError_t err;
    err = hipHostRegister(ptr, size, hipHostRegisterPortable);
    return (int) err;
}

int cudaHostUnregisterWrapper(void* ptr){
    hipError_t err;
    err = hipHostUnregister(ptr);
    return (int) err;
}

int cudaGetDeviceCountWrapper(int *count){
    return (int) hipGetDeviceCount(count);
}




