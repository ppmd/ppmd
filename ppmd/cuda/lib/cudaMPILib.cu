#include "hip/hip_runtime.h"
#include "cudaMPILib.h"



// DIrectly based on MPI

int MPIErrorCheck_cuda(const int error_code){

    int err = 0;
    if (error_code != MPI_SUCCESS) {

       char error_string[BUFSIZ];
       int length_of_error_string;

       MPI_Error_string(error_code, error_string, &length_of_error_string);
       //fprintf(stderr, "%3d: %s\n", my_rank, error_string);
       cout << error_string << endl;

       err = 1;
    }

    return err;
}





int MPI_Bcast_cuda(const int FCOMM, void* buffer, const int byte_count, const int root){
    MPI_Comm COMM = MPI_Comm_f2c(FCOMM);

    MPI_Errhandler_set(COMM, MPI_ERRORS_RETURN);
    const int err = MPI_Bcast( buffer,
                               byte_count,
                               MPI_BYTE,
                               root,
                               COMM
                             );

    return err;
}


int MPI_Gatherv_cuda(const int FCOMM,
                     const void* s_buffer,
                     const int s_count,
                     void* r_buffer,
                     const int* r_counts,
                     const int* r_disps,
                     const int root
                     ){

    MPI_Comm COMM = MPI_Comm_f2c(FCOMM);

    MPI_Errhandler_set(COMM, MPI_ERRORS_RETURN);
    const int err = MPI_Gatherv(s_buffer,
                                s_count,
                                MPI_BYTE,
                                r_buffer,
                                r_counts,
                                r_disps,
                                MPI_BYTE,
                                root,
                                COMM);

    return err;
}














// MPI related

namespace _cudaFindNewSlots
{
    __constant__ int d_n1;
    __constant__ int d_n2;


    __global__ void cudaFindNewSlots_kernel1(const int * __restrict__ d_scan,
                                             int * __restrict__ d_empties ){

        int _ix = threadIdx.x + blockIdx.x*blockDim.x;
        if (_ix < d_n1){
            if (d_scan[_ix] < d_scan[_ix+1]){
                 d_empties[d_scan[_ix]] = _ix;
            }
        }
        return;
    }


    __global__ void cudaFindNewSlots_kernel2(const int * __restrict__ d_scan,
                                             int * __restrict__ d_sources){

        int _ix = threadIdx.x + blockIdx.x*blockDim.x;

        if (_ix < d_n2){

            const int ix = _ix + d_n1;
            const int num_empty_after_end = d_scan[ix] - d_scan[d_n1];

            //if non empty
            if (d_scan[ix] == d_scan[ix+1]){
                d_sources[_ix-num_empty_after_end] = ix;
            }

        }
        return;
    }



}


int cudaFindEmptySlots(const int blocksize1[3],
                       const int threadsize1[3],
                       const int* d_scan,
                       const int h_n1,
                       int * d_empties
                       ){

    //device constant copy.
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_cudaFindNewSlots::d_n1), &h_n1, sizeof(int)));


    dim3 bs; bs.x = blocksize1[0]; bs.y = blocksize1[1]; bs.z = blocksize1[2];
    dim3 ts; ts.x = threadsize1[0]; ts.y = threadsize1[1]; ts.z = threadsize1[2];

    _cudaFindNewSlots::cudaFindNewSlots_kernel1<<<bs,ts>>>(d_scan, d_empties);

    return (int) hipDeviceSynchronize();
}



int cudaFindNewSlots(const int blocksize2[3],
                     const int threadsize2[3],
                     const int* d_scan,
                     const int h_n1,
                     const int h_n2,
                     int * d_sources
                     ){

    //device constant copy.
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_cudaFindNewSlots::d_n1), &h_n1, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_cudaFindNewSlots::d_n2), &h_n2, sizeof(int)));

    dim3 bs2; bs2.x = blocksize2[0]; bs2.y = blocksize2[1]; bs2.z = blocksize2[2];
    dim3 ts2; ts2.x = threadsize2[0]; ts2.y = threadsize2[1]; ts2.z = threadsize2[2];

    _cudaFindNewSlots::cudaFindNewSlots_kernel2<<<bs2,ts2>>>(d_scan, d_sources);

    return (int) hipDeviceSynchronize();

}





hipError_t cudaCreateLaunchArgs(
        const int N,    // Total minimum number of threads.
        const int Nt,   // Number of threads per block.
        dim3* bs,       // RETURN: grid of thread blocks.
        dim3* ts        // RETURN: grid of threads
        ){

    if ((N<0) || (Nt<0)){
        cout << "cudaCreateLaunchArgs Error: Invalid desired number of total threads " << N << 
            "or invalid number of threads per block " << Nt << endl;
        return hipErrorUnknown;
    }
    
    const int Nb = ceil(((double) N) / ((double) Nt));

    bs->x = Nb; bs->y = 1; bs->z = 1;
    ts->x = Nt; ts->y = 1; ts->z = 1;

    return hipSuccess;
}





/*
namespace _ExSizes
{   

    __global__ void GatherCellCounts(
        const int n,                        // Number of cells to inspect
        const int* __restrict__ D_b_arr,    // actual indices of boundary cells
        int* D_b_tmp,                       // space to place boundary cell counts
        int* D_tmp_count                    // reduce the count accross cells into here
        ){


        int _ix = threadIdx.x + blockIdx.x*blockDim.x;
        int tt = 0;


        if (_ix < n){

            D_b_tmp[_ix] = D_b_arr[_ix];   // copy this cell count
            tt = D_b_arr[_ix];             // set local tt to this cell count

        }

        // reduce tmp count accross warp
        cc = warpReduceSum(cc);

        // reduce into global mem
        if (threadIdx.x == 0){
            atomicAdd(D_tmp_count, cc);
        }

        return;
    }






}

int cudaExchangeCellCounts(
        const int FCOMM,                        // Fortran communicator
        const int* __restrict__ H_SEND_RANKS,   // send ranks
        const int* __restrict__ H_RECV_RANKS,   // recv ranks 
        const int* __restrict__ H_h_ind,        // The starting indices for the halo cells
        const int* __restrict__ H_b_ind,        // The starting indices for the bound cellsi
        const int* __restrict__ D_h_arr,        // The halo cell indices
        const int* __restrict__ D_b_arr,        // The boundary cell indices
        const int* __restrict__ D_CCC,          // Cell contents count array
        int* __restrict__ H_halo_count,         // RETURN: Number of halo particles
        int* __restrict__ H_tmp_count,          // RETURN: Amount of temporary space needed
        int* __restrict__ D_h_tmp,              // Temp storage for halo counts
        int* __restrict__ H_b_tmp,              // Temp storage for bundary counts
        int* __restrict__ H_dir_counts          // RETURN: Total expected recv counts per dir
        )
{   

    // var to use for errors
    int err;

    // vars for blocks and threads
    dim3 bs, ts;

    // MPI initialisations
    MPI_Comm COMM = MPI_Comm_f2c(FCOMM);
    int rank; MPI_Comm_rank(COMM, &rank);
    MPI_Status MPI_STATUS;

    
    //reset the return counts
    *H_tmp_count = 0;
    *H_halo_count = 0;

    int * D_tmp_count;
    
    // make a device tmp
    err = (int) hipMalloc(&D_tmp_count, sizeof(int)); 
    if (err != 0) { return err; }
    
    // ensure is zero
    err = (int) hipMemcpy(D_tmp_count, H_tmp_count, sizeof(int), hipMemcpyHostToDevice);
    if (err != 0) { return err; }



    for(int dir=0 ; dir<6 ; dir++ ){

        // Here we want to collect the local cell counts for a direction on the device
        // exchange these sizes and get the total for the direction
    
        const int dir_cell_count = H_b_ind[dir+1] - H_b_ind[dir];
        err = cudaCreateLaunchArgs(dir_cell_count, 256, &bs, &ts);
        if (err != 0) { return err; }
        
        _ExSizes::GatherCellCounts<<<bs,ts>>>(dir_cell_count,
                                              D_b_arr+H_b_ind[dir],
                                              D_b_tmp,
                                              D_tmp_count);
        
        err = hipDeviceSynchronize();
        if (err != 0) { return err; }

    }


    return 0;
}

*/

















