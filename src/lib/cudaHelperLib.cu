#include "cudaHelperLib.h"

int cudaErrorCheck(int err){
    checkCudaErrors((hipError_t) err);
    return err;
}

void cudaCpyHostToDevice(void* dst, const void* src, size_t count){
    checkCudaErrors(hipMemcpy(dst,src,count,hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}

void cudaCpyDeviceToHost(void* dst, const void* src, size_t count){
    checkCudaErrors(hipMemcpy(dst,src,count,hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());
    return;
}
